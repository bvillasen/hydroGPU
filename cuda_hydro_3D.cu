#include "hip/hip_runtime.h"
#include <pycuda-helpers.hpp>

#define N_W N_WIDTH
#define N_H N_HEIGHT
#define N_D N_DEPTH

extern "C"   // ensure functions name to be exactly the same as below
{
	__global__ void convertToUCHAR( const int field, const int nCells, cudaP normaliztion, cudaP *values, unsigned char *psiUCHAR ){
		int tid = blockIdx.x*blockDim.x + threadIdx.x;
		psiUCHAR[tid] = (unsigned char) ( -255*( values[field*nCells + tid]*normaliztion -1 ));
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__ void reduction_max_kernel( double *input, double *output ){
		__shared__ double sh_data[512];

		unsigned int tid = threadIdx.x;
		unsigned int i   = blockIdx.x * blockDim.x  + threadIdx.x;
		sh_data[tid] = max( input[i], input[i + blockDim.x*gridDim.x ] ) ;
		__syncthreads();

		for( unsigned int s = blockDim.x/2; s>0; s >>= 1){
			if ( tid < s ) sh_data[tid] = max( sh_data[tid], sh_data[tid+s] );
			__syncthreads();
		}

		if ( tid == 0 ) output[ blockIdx.x ] = sh_data[0];
	}
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__ void reduction_min_kernel( double *input, double *output ){
		__shared__ double sh_data[512];

		unsigned int tid = threadIdx.x;
		unsigned int i   = blockIdx.x * blockDim.x  + threadIdx.x;
		sh_data[tid] = min( input[i], input[i + blockDim.x*gridDim.x ] ) ;
		__syncthreads();

		for( unsigned int s = blockDim.x/2; s>0; s >>= 1){
			if ( tid < s ) sh_data[tid] = min( sh_data[tid], sh_data[tid+s] );
			__syncthreads();
		}

		if ( tid == 0 ) output[ blockIdx.x ] = sh_data[0];
	}
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__ void writeBound(  const int boundAxis, const int nCells, double *cnsv, double *bound,
                  const int t_j, const int t_i, const int t_k, const int tid ){
    int boundId, stride;
    if ( boundAxis == 1 ){    //X BOUNDERIES
			boundId = t_i + t_k*N_H;
			stride = N_H*N_D;
		}
    if ( boundAxis == 2 ) {   //Y BOUNDERIES
			boundId = t_j + t_k*N_W;
			stride = N_W*N_D;
		}
    if ( boundAxis == 3 ) {  //Z BOUNDERIES
			boundId = t_j + t_i*N_W;
			stride = N_W*N_H;
		}
    bound[0*stride + boundId] = cnsv[0*nCells + tid];
    bound[1*stride + boundId] = cnsv[1*nCells + tid];
    bound[2*stride + boundId] = cnsv[2*nCells + tid];
    bound[3*stride + boundId] = cnsv[3*nCells + tid];
    bound[4*stride + boundId] = cnsv[4*nCells + tid];
  }

  __global__ void setBounderies( const int nCells, double *cnsv,
				 double* bound_l, double* bound_r, double* bound_d, double* bound_u, double* bound_b, double *bound_t ){
    int t_j = blockIdx.x*blockDim.x + threadIdx.x;
    int t_i = blockIdx.y*blockDim.y + threadIdx.y;
    int t_k = blockIdx.z*blockDim.z + threadIdx.z;
    int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;

    bool boundBlock = false;
    if ( blockIdx.x==0 || blockIdx.y==0 || blockIdx.z==0 ) boundBlock = true;
    if ( blockIdx.x==(gridDim.x-1) || blockIdx.y==(gridDim.y-1) || blockIdx.z==(gridDim.z-1) ) boundBlock = true;

    if ( !boundBlock ) return;

    if ( t_j==0 )       writeBound( 1, nCells, cnsv, bound_l, t_j, t_i, t_k, tid );
    if ( t_j==(N_W-1) ) writeBound( 1, nCells, cnsv, bound_r, t_j, t_i, t_k, tid );

    if ( t_i==0 )       writeBound( 2, nCells, cnsv, bound_d, t_j, t_i, t_k, tid );
    if ( t_i==(N_H-1) ) writeBound( 2, nCells, cnsv, bound_u, t_j, t_i, t_k, tid );

    if ( t_k==0 )       writeBound( 3, nCells, cnsv, bound_b, t_j, t_i, t_k, tid );
    if ( t_k==(N_D-1) ) writeBound( 3, nCells, cnsv, bound_t, t_j, t_i, t_k, tid );
  }

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	__device__ double hll_interFlux( double val_l, double val_r, double F_l, double F_r, double s_l, double s_r ){
	  if ( s_l > 0 ) return F_l;
	  if ( s_r < 0 ) return F_r;
	  return ( s_r*F_l - s_l*F_r + s_l*s_r*( val_r - val_l ) ) / ( s_r - s_l );
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	__device__ void writeInterFlux(const int coord, const int writeStride, int tid,
					double rho_l, double rho_r, double vx_l, double vx_r, double vy_l, double vy_r, double vz_l, double vz_r, double E_l, double E_r,
					double p_l, double p_r, double s_l, double s_r,
					double *iFlx ){

		// Adjacent fluxes from left and center cell
		double F_l, F_r;

		//iFlx rho
		if ( coord == 1 ){
			F_l = rho_l * vx_l;
			F_r = rho_r * vx_r;
		}
		else if ( coord == 2 ){
			F_l = rho_l * vy_l;
			F_r = rho_r * vy_r;
		}
		else if ( coord == 3 ){
			F_l = rho_l * vz_l;
			F_r = rho_r * vz_r;
		}
		iFlx[0*writeStride + tid] = hll_interFlux( rho_l, rho_r, F_l, F_r, s_l, s_r );

		//iFlx rho * vx
		if ( coord == 1 ){
			F_l = rho_l * vx_l * vx_l + p_l;
			F_r = rho_r * vx_r * vx_r + p_r;
		}
		else if ( coord == 2 ){
			F_l = rho_l * vx_l * vy_l;
			F_r = rho_r * vx_r * vy_r;
		}
		else if ( coord == 3 ){
			F_l = rho_l * vx_l * vz_l;
			F_r = rho_r * vx_r * vz_r;
		}
		iFlx[1*writeStride + tid] = hll_interFlux( rho_l*vx_l, rho_r*vx_r, F_l, F_r, s_l, s_r );

		//iFlx rho * vy
		if ( coord == 1 ){
			F_l = rho_l * vy_l * vx_l ;
			F_r = rho_r * vy_r * vx_r ;
		}
		else if ( coord == 2 ){
			F_l = rho_l * vy_l * vy_l + p_l;
			F_r = rho_r * vy_r * vy_r + p_r;
		}
		else if ( coord == 3 ){
			F_l = rho_l * vy_l * vz_l;
			F_r = rho_r * vy_r * vz_r;
		}
		iFlx[2*writeStride + tid] = hll_interFlux( rho_l*vy_l, rho_r*vy_r, F_l, F_r, s_l, s_r );

		//iFlx rho * vz
		if ( coord == 1 ){
			F_l = rho_l * vz_l * vx_l ;
			F_r = rho_r * vz_r * vx_r ;
		}
		else if ( coord == 2 ){
			F_l = rho_l * vz_l * vy_l ;
			F_r = rho_r * vz_r * vy_r ;
		}
		else if ( coord == 3 ){
			F_l = rho_l * vz_l * vz_l + p_l ;
			F_r = rho_r * vz_r * vz_r + p_r ;
		}
		iFlx[3*writeStride + tid] = hll_interFlux( rho_l*vz_l, rho_r*vz_r, F_l, F_r, s_l, s_r );

		//iFlx E
		if ( coord == 1 ){
			F_l = vx_l * ( E_l + p_l ) ;
			F_r = vx_r * ( E_r + p_r ) ;
		}
		else if ( coord == 2 ){
			F_l = vy_l * ( E_l + p_l ) ;
			F_r = vy_r * ( E_r + p_r ) ;
		}
		else if ( coord == 3 ){
			F_l = vz_l * ( E_l + p_l ) ;
			F_r = vz_r * ( E_r + p_r ) ;
		}
		iFlx[4*writeStride + tid] = hll_interFlux( E_l, E_r, F_l, F_r, s_l, s_r );
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  __global__ void setInterFlux_hll( const int coord, const int nCells, const double gamma, const double dx, const double dy, const double dz,
				 double *cnsv, double *iFlx,
				 double *bound_l, double *bound_r,
				 double *iFlx_bnd,
         double* times ){
    int t_j = blockIdx.x*blockDim.x + threadIdx.x;
    int t_i = blockIdx.y*blockDim.y + threadIdx.y;
    int t_k = blockIdx.z*blockDim.z + threadIdx.z;
    int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;

    int tid_adj, boundId, stride;
    double v2;
    double rho_l, vx_l, vy_l, vz_l, E_l, p_l;
    double rho_c, vx_c, vy_c, vz_c, E_c, p_c;

    //Set adjacent id
    if ( coord == 1 ){
      if ( t_j == 0) tid_adj = (t_j) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
      else tid_adj = (t_j-1) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    }
    if ( coord == 2 ){
      if ( t_i == 0) tid_adj = t_j + (t_i)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
      else tid_adj = t_j + (t_i-1)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    }
    if ( coord == 3 ){
      if ( t_k == 0) tid_adj = t_j + t_i*blockDim.x*gridDim.x + (t_k)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
      else tid_adj = t_j + t_i*blockDim.x*gridDim.x + (t_k-1)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    }
    //Read adjacent and center conservatives
    rho_l = cnsv[ 0*nCells + tid_adj ];
    rho_c = cnsv[ 0*nCells + tid ];

    vx_l = cnsv[ 1*nCells + tid_adj ] / rho_l;
    vx_c = cnsv[ 1*nCells + tid ] / rho_c;

    vy_l = cnsv[ 2*nCells + tid_adj ] / rho_l;
    vy_c = cnsv[ 2*nCells + tid ] / rho_c;

    vz_l = cnsv[ 3*nCells + tid_adj ] / rho_l;
    vz_c = cnsv[ 3*nCells + tid ] / rho_c;

    E_l = cnsv[ 4*nCells + tid_adj ];
    E_c = cnsv[ 4*nCells + tid ];

    //Load and apply boundery conditions
    if ( coord == 1 ){
      boundId = t_i + t_k*N_H;
			stride = N_H*N_D;
      if ( t_j == 0) {
        rho_l = bound_l[0*stride + boundId];
        vx_l  = bound_l[1*stride + boundId] / rho_l;
        vy_l  = bound_l[2*stride + boundId] / rho_l;
        vz_l  = bound_l[3*stride + boundId] / rho_l;
        E_l   = bound_l[4*stride + boundId];
      }
    }
    if ( coord == 2 ){
      boundId = t_j + t_k*N_W;
			stride = N_W*N_D;
      if ( t_i == 0) {
        rho_l = bound_l[0*stride + boundId];
        vx_l  = bound_l[1*stride + boundId] / rho_l;
        vy_l  = bound_l[2*stride + boundId] / rho_l;
        vz_l  = bound_l[3*stride + boundId] / rho_l;
        E_l   = bound_l[4*stride + boundId];
      }
    }
    if ( coord == 3 ){
      boundId = t_j + t_i*N_W;
			stride = N_W*N_H;
      if ( t_k == 0) {
        rho_l = bound_l[0*stride + boundId];
        vx_l  = bound_l[1*stride + boundId] / rho_l;
        vy_l  = bound_l[2*stride + boundId] / rho_l;
        vz_l  = bound_l[3*stride + boundId] / rho_l;
        E_l   = bound_l[4*stride + boundId];
      }
    }


    //Boundary bounce condition
      if ( coord==1 && t_j == 0 ) vx_l = -vx_c;
        //Boundary bounce condition
      if ( coord==2 && t_i == 0 ) vy_l = -vy_c;
      //Boundary bounce condition
      if ( coord==3 && t_k == 0 ) vz_l = -vz_c;

    v2    = vx_l*vx_l + vy_l*vy_l + vz_l*vz_l;
    p_l   = ( E_l - rho_l*v2/2 ) * (gamma-1);

    v2    = vx_c*vx_c + vy_c*vy_c + vz_c*vz_c;
    p_c   = ( E_c - rho_c*v2/2 ) * (gamma-1);

    double cs_l, cs_c, s_l, s_c;
    cs_l = sqrt( p_l * gamma / rho_l );
    cs_c = sqrt( p_c * gamma / rho_c );

    if ( coord == 1 ){
      s_l = min( vx_l - cs_l, vx_c - cs_c );
      s_c = max( vx_l + cs_l, vx_c + cs_c );
      //Use v2 to save time minimum
      v2 = dx / ( abs( vx_c ) + cs_c );
      v2 = min( v2, dy / ( abs( vy_c ) + cs_c ) );
      v2 = min( v2, dz / ( abs( vz_c ) + cs_c ) );
      times[ tid ] = v2;
    }

    else if ( coord == 2 ){
      s_l = min( vy_l - cs_l, vy_c - cs_c );
      s_c = max( vy_l + cs_l, vy_c + cs_c ); 
    }

    else if ( coord == 3 ){
      s_l = min( vz_l - cs_l, vz_c - cs_c );
      s_c = max( vz_l + cs_l, vz_c + cs_c );
    }

    writeInterFlux( coord, nCells, tid, rho_l, rho_c, vx_l, vx_c, vy_l, vy_c, vz_l, vz_c, E_l, E_c,
            p_l, p_c, s_l, s_c, iFlx  );

    //Get iFlux_r for most right cell
    // if ( blockIdx.x!=(gridDim.x-1) || blockIdx.y!=(gridDim.y-1) || blockIdx.z!=(gridDim.z-1) ) return;

    if ( coord == 1 ){
      if ( t_j != (N_W-1) ) return;
    }
    if ( coord == 2 ){
      if ( t_i != (N_H-1) ) return;
    }
    if ( coord == 3 ){
      if ( t_k != (N_D-1) ) return;
    }

    rho_l = rho_c;
    vx_l = vx_c;
    vy_l = vy_c;
    vz_l = vz_c;
    E_l = E_c;
    p_l = p_c;
    cs_l = cs_c;



    //Load Bounderies for right part of the box_size
    rho_c = bound_r[0*stride + boundId];
    vx_c  = bound_r[1*stride + boundId] / rho_c;
    vy_c  = bound_r[2*stride + boundId] / rho_c;
    vz_c  = bound_r[3*stride + boundId] / rho_c;
    E_c   = bound_r[4*stride + boundId];

    // //Boundary bounce conditions
    if ( coord==1 && t_j == (N_W-1) ) vx_c = -vx_l;
      //Boundary bounce condition
    if ( coord==2 && t_i == (N_H-1) ) vy_c = -vy_l;
    //Boundary bounce condition
    if ( coord==3 && t_k == (N_D-1) ) vz_c = -vz_l;

    v2    = vx_c*vx_c + vy_c*vy_c + vz_c*vz_c;
    p_c   = ( E_c - rho_c*v2/2 ) * (gamma-1);
    cs_c = sqrt( p_c * gamma / rho_c );
    if ( coord == 1 ){
      s_l = min( vx_l - cs_l, vx_c - cs_c );
      s_c = max( vx_l + cs_l, vx_c + cs_c );
    }

    else if ( coord == 2 ){
      s_l = min( vy_l - cs_l, vy_c - cs_c );
      s_c = max( vy_l + cs_l, vy_c + cs_c );
    }

    else if ( coord == 3 ){
      s_l = min( vz_l - cs_l, vz_c - cs_c );
      s_c = max( vz_l + cs_l, vz_c + cs_c );
    }
		writeInterFlux( coord, stride, boundId, rho_l, rho_c, vx_l, vx_c, vy_l, vy_c, vz_l, vz_c, E_l, E_c,
            p_l, p_c, s_l, s_c, iFlx_bnd  );
    // writeInterFlux_b( coord, boundId, rho_l, rho_c, vx_l, vx_c, vy_l, vy_c, vz_l, vz_c, E_l, E_c,
    //         p_l, p_c, s_l, s_c, iFlx, iFlx_1_bnd, iFlx_2_bnd, iFlx_3_bnd, iFlx_4_bnd, iFlx_5_bnd  );
  }
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	__global__ void getInterFlux_hll( const int coord, const int nCells, const double dt,  const double gamma,
				 const double dx, const double dy, const double dz,
				 double *cnsv_adv, double *iFlx, double *iFlx_bnd ){
				 //  double* gForceX, double* gForceY, double* gForceZ, double* gravWork ){
		int t_j = blockIdx.x*blockDim.x + threadIdx.x;
		int t_i = blockIdx.y*blockDim.y + threadIdx.y;
		int t_k = blockIdx.z*blockDim.z + threadIdx.z;
		int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;

		int tid_adj, boundId, stride;
		double iFlx1_l, iFlx2_l, iFlx3_l, iFlx4_l, iFlx5_l;
		double iFlx1_r, iFlx2_r, iFlx3_r, iFlx4_r, iFlx5_r;
		double delta;

		//Set adjacent id
		if ( coord == 1 ){
			if ( t_j == N_W-1 ) tid_adj = (t_j) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
			else tid_adj = (t_j+1) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
			delta = dt / dx;
		}
		if ( coord == 2 ){
			if ( t_i == N_H-1 ) tid_adj = t_j + (t_i)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
			else tid_adj = t_j + (t_i+1)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
			delta = dt / dy;
		}
		if ( coord == 3 ){
			if ( t_k == N_D-1) tid_adj = t_j + t_i*blockDim.x*gridDim.x + (t_k)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
			else tid_adj = t_j + t_i*blockDim.x*gridDim.x + (t_k+1)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
			delta = dt / dz;
		}

		//Read inter-cell fluxes
		iFlx1_l = iFlx[0*nCells + tid ];
		iFlx1_r = iFlx[0*nCells + tid_adj ];

		iFlx2_l = iFlx[1*nCells + tid ];
		iFlx2_r = iFlx[1*nCells + tid_adj ];

		iFlx3_l = iFlx[2*nCells + tid ];
		iFlx3_r = iFlx[2*nCells + tid_adj ];

		iFlx4_l = iFlx[3*nCells + tid ];
		iFlx4_r = iFlx[3*nCells + tid_adj ];

		iFlx5_l = iFlx[4*nCells + tid ];
		iFlx5_r = iFlx[4*nCells + tid_adj ];

		if ( coord == 1 ){
			boundId = t_i + t_k*N_H;
			stride = N_H*N_D;
			if ( t_j == (N_W-1) ) {
				iFlx1_r = iFlx_bnd[0*stride + boundId];
				iFlx2_r = iFlx_bnd[1*stride + boundId];
				iFlx3_r = iFlx_bnd[2*stride + boundId];
				iFlx4_r = iFlx_bnd[3*stride + boundId];
				iFlx5_r = iFlx_bnd[4*stride + boundId];
			}
		}
		if ( coord == 2 ){
			boundId = t_j + t_k*N_W;
			stride = N_W*N_D;
			if ( t_i == (N_H-1) ) {
				iFlx1_r = iFlx_bnd[0*stride + boundId];
				iFlx2_r = iFlx_bnd[1*stride + boundId];
				iFlx3_r = iFlx_bnd[2*stride + boundId];
				iFlx4_r = iFlx_bnd[3*stride + boundId];
				iFlx5_r = iFlx_bnd[4*stride + boundId];
			}
		}
		if ( coord == 3 ){
			boundId = t_j + t_i*N_W;
			stride = N_W*N_H;
			if ( t_k == (N_D-1) ) {
				iFlx1_r = iFlx_bnd[0*stride + boundId];
				iFlx2_r = iFlx_bnd[1*stride + boundId];
				iFlx3_r = iFlx_bnd[2*stride + boundId];
				iFlx4_r = iFlx_bnd[3*stride + boundId];
				iFlx5_r = iFlx_bnd[4*stride + boundId];
			}
		}
		//Load and apply boundery conditions
		// if ( coord == 1 ){
		//   boundId = t_i + t_k*N_H;
		//   if ( t_j == (N_W-1) ) {
		//     iFlx1_r = iFlx_1_bnd[boundId];
		//     iFlx2_r = iFlx_2_bnd[boundId];
		//     iFlx3_r = iFlx_3_bnd[boundId];
		//     iFlx4_r = iFlx_4_bnd[boundId];
		//     iFlx5_r = iFlx_5_bnd[boundId];
		//   }
		// }
		// if ( coord == 2 ){
		//   boundId = t_j + t_k*N_W;
		//   if ( t_i == (N_H-1) ) {
		//     iFlx1_r = iFlx_1_bnd[boundId];
		//     iFlx2_r = iFlx_2_bnd[boundId];
		//     iFlx3_r = iFlx_3_bnd[boundId];
		//     iFlx4_r = iFlx_4_bnd[boundId];
		//     iFlx5_r = iFlx_5_bnd[boundId];
		//   }
		// }
		// if ( coord == 3 ){
		//   boundId = t_j + t_i*N_W;
		//   if ( t_k == (N_D-1) ) {
		//     iFlx1_r = iFlx_1_bnd[boundId];
		//     iFlx2_r = iFlx_2_bnd[boundId];
		//     iFlx3_r = iFlx_3_bnd[boundId];
		//     iFlx4_r = iFlx_4_bnd[boundId];
		//     iFlx5_r = iFlx_5_bnd[boundId];
		//   }
		// }

		//Advance the consv values
		// cnsv_1[ tid ] = cnsv_1[ tid ] - delta*( iFlx1_r - iFlx1_l );
		// cnsv_2[ tid ] = cnsv_2[ tid ] - delta*( iFlx2_r - iFlx2_l ) + dt*gForceX[tid]*50;
		// cnsv_3[ tid ] = cnsv_3[ tid ] - delta*( iFlx3_r - iFlx3_l ) + dt*gForceY[tid]*50;
		// cnsv_4[ tid ] = cnsv_4[ tid ] - delta*( iFlx4_r - iFlx4_l ) + dt*gForceZ[tid]*50;
		// cnsv_5[ tid ] = cnsv_5[ tid ] - delta*( iFlx5_r - iFlx5_l ) + dt*gravWork[tid]*50;

		if ( coord == 1 ){
			cnsv_adv[0*nCells +  tid ] = -delta*( iFlx1_r - iFlx1_l );
			cnsv_adv[1*nCells +  tid ] = -delta*( iFlx2_r - iFlx2_l );
			cnsv_adv[2*nCells +  tid ] = -delta*( iFlx3_r - iFlx3_l );
			cnsv_adv[3*nCells +  tid ] = -delta*( iFlx4_r - iFlx4_l );
			cnsv_adv[4*nCells +  tid ] = -delta*( iFlx5_r - iFlx5_l );
		}
		else{
			cnsv_adv[0*nCells +  tid ] = cnsv_adv[0*nCells +  tid ] - delta*( iFlx1_r - iFlx1_l );
			cnsv_adv[1*nCells +  tid ] = cnsv_adv[1*nCells +  tid ] - delta*( iFlx2_r - iFlx2_l );
			cnsv_adv[2*nCells +  tid ] = cnsv_adv[2*nCells +  tid ] - delta*( iFlx3_r - iFlx3_l );
			cnsv_adv[3*nCells +  tid ] = cnsv_adv[3*nCells +  tid ] - delta*( iFlx4_r - iFlx4_l );
			cnsv_adv[4*nCells +  tid ] = cnsv_adv[4*nCells +  tid ] - delta*( iFlx5_r - iFlx5_l );
		}
	}

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	__global__ void addDtoD( const int nCells,
			double *dst, double *sum ){
		int t_j = blockIdx.x*blockDim.x + threadIdx.x;
		int t_i = blockIdx.y*blockDim.y + threadIdx.y;
		int t_k = blockIdx.z*blockDim.z + threadIdx.z;
		int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;

		dst[0*nCells + tid] = dst[0*nCells + tid] + sum[0*nCells + tid];
		dst[1*nCells + tid] = dst[1*nCells + tid] + sum[1*nCells + tid];
		dst[2*nCells + tid] = dst[2*nCells + tid] + sum[2*nCells + tid];
		dst[3*nCells + tid] = dst[3*nCells + tid] + sum[3*nCells + tid];
		dst[4*nCells + tid] = dst[4*nCells + tid] + sum[4*nCells + tid];
	}
}//End of extern 'C'

















//
// //Textures for conserv
// texture< fp_tex_cudaP, hipTextureType3D, hipReadModeElementType> tex_1;
// texture< fp_tex_cudaP, hipTextureType3D, hipReadModeElementType> tex_2;
// texture< fp_tex_cudaP, hipTextureType3D, hipReadModeElementType> tex_3;
// texture< fp_tex_cudaP, hipTextureType3D, hipReadModeElementType> tex_4;
// texture< fp_tex_cudaP, hipTextureType3D, hipReadModeElementType> tex_5;
//
//
// //Surfaces for Fluxes
// surface< void, cudaSurfaceType3D> surf_1;
// surface< void, cudaSurfaceType3D> surf_2;
// surface< void, cudaSurfaceType3D> surf_3;
// surface< void, cudaSurfaceType3D> surf_4;
// surface< void, cudaSurfaceType3D> surf_5;
//
// __global__ void setInterFlux_hll( const int coord, const cudaP gamma, const cudaP dx, const cudaP dy, const cudaP dz,
// 			 cudaP* cnsv_1, cudaP* cnsv_2, cudaP* cnsv_3, cudaP* cnsv_4, cudaP* cnsv_5,
// 			 float* times ){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int t_k = blockIdx.z*blockDim.z + threadIdx.z;
//   int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//
//   cudaP v2;
//   cudaP rho_l, vx_l, vy_l, vz_l, E_l, p_l;
//   cudaP rho_c, vx_c, vy_c, vz_c, E_c, p_c;
// //   float time;
//   //Read adjacent conserv
//   if ( coord == 1 ){
//     rho_l = fp_tex3D( tex_1, t_j-1, t_i, t_k);
//     rho_c = fp_tex3D( tex_1, t_j, t_i, t_k);
//
//     vx_l  = fp_tex3D( tex_2, t_j-1, t_i, t_k) / rho_l;
//     vx_c  = fp_tex3D( tex_2, t_j, t_i, t_k)   / rho_c;
//
//     vy_l  = fp_tex3D( tex_3, t_j-1, t_i, t_k) / rho_l;
//     vy_c  = fp_tex3D( tex_3, t_j, t_i, t_k)   / rho_c;
//
//     vz_l  = fp_tex3D( tex_4, t_j-1, t_i, t_k) / rho_l;
//     vz_c  = fp_tex3D( tex_4, t_j, t_i, t_k)   / rho_c;
//
//     E_l   = fp_tex3D( tex_5, t_j-1, t_i, t_k);
//     E_c   = fp_tex3D( tex_5, t_j, t_i, t_k);
//
//
//   }
//   else if ( coord == 2 ){
//     rho_l = fp_tex3D( tex_1, t_j, t_i-1, t_k);
//     rho_c = fp_tex3D( tex_1, t_j, t_i, t_k);
//
//     vx_l  = fp_tex3D( tex_2, t_j, t_i-1, t_k) / rho_l;
//     vx_c  = fp_tex3D( tex_2, t_j, t_i, t_k)   / rho_c;
//
//     vy_l  = fp_tex3D( tex_3, t_j, t_i-1, t_k) / rho_l;
//     vy_c  = fp_tex3D( tex_3, t_j, t_i, t_k)   / rho_c;
//
//     vz_l  = fp_tex3D( tex_4, t_j, t_i-1, t_k) / rho_l;
//     vz_c  = fp_tex3D( tex_4, t_j, t_i, t_k)   / rho_c;
//
//     E_l   = fp_tex3D( tex_5, t_j, t_i-1, t_k);
//     E_c   = fp_tex3D( tex_5, t_j, t_i, t_k);
//
//
//   }
//   else if ( coord == 3 ){
//     rho_l = fp_tex3D( tex_1, t_j, t_i, t_k-1);
//     rho_c = fp_tex3D( tex_1, t_j, t_i, t_k);
//
//     vx_l  = fp_tex3D( tex_2, t_j, t_i, t_k-1) / rho_l;
//     vx_c  = fp_tex3D( tex_2, t_j, t_i, t_k)   / rho_c;
//
//     vy_l  = fp_tex3D( tex_3, t_j, t_i, t_k-1) / rho_l;
//     vy_c  = fp_tex3D( tex_3, t_j, t_i, t_k)   / rho_c;
//
//     vz_l  = fp_tex3D( tex_4, t_j, t_i, t_k-1) / rho_l;
//     vz_c  = fp_tex3D( tex_4, t_j, t_i, t_k)   / rho_c;
//
//     E_l   = fp_tex3D( tex_5, t_j, t_i, t_k-1);
//     E_c   = fp_tex3D( tex_5, t_j, t_i, t_k);
//
//
//   }
// //   //Boundary bounce condition
// //     if ( t_j == 0 ) vx_l = -vx_c;
// //       //Boundary bounce condition
// //     if ( t_i == 0 ) vy_l = -vy_c;
// //     //Boundary bounce condition
// //     if ( t_k == 0 ) vz_l = -vz_c;
//
//   v2    = vx_l*vx_l + vy_l*vy_l + vz_l*vz_l;
//   p_l   = ( E_l - rho_l*v2/2 ) * (gamma-1);
//
//   v2    = vx_c*vx_c + vy_c*vy_c + vz_c*vz_c;
//   p_c   = ( E_c - rho_c*v2/2 ) * (gamma-1);
//
//
//   cudaP cs_l, cs_c, s_l, s_c;
//   cs_l = sqrt( p_l * gamma / rho_l );
//   cs_c = sqrt( p_c * gamma / rho_c );
//
//
//
//   if ( coord == 1 ){
//     s_l = min( vx_l - cs_l, vx_c - cs_c );
//     s_c = max( vx_l + cs_l, vx_c + cs_c );
//     //Use v2 to save time minimum
//     v2 = dx / ( abs( vx_c ) + cs_c );
//     v2 = min( v2, dy / ( abs( vy_c ) + cs_c ) );
//     v2 = min( v2, dz / ( abs( vz_c ) + cs_c ) );
//     times[ tid ] = v2;
//   }
//   else if ( coord == 2 ){
//     s_l = min( vy_l - cs_l, vy_c - cs_c );
//     s_c = max( vy_l + cs_l, vy_c + cs_c );
//   }
//   else if ( coord == 3 ){
//     s_l = min( vz_l - cs_l, vz_c - cs_c );
//     s_c = max( vz_l + cs_l, vz_c + cs_c );
//   }
//
//   // Adjacent fluxes from left and center cell
//   cudaP F_l, F_c, iFlx;
//
//   //iFlx rho
//   if ( coord == 1 ){
//     F_l = rho_l * vx_l;
//     F_c = rho_c * vx_c;
//   }
//   else if ( coord == 2 ){
//     F_l = rho_l * vy_l;
//     F_c = rho_c * vy_c;
//   }
//   else if ( coord == 3 ){
//     F_l = rho_l * vz_l;
//     F_c = rho_c * vz_c;
//   }
//   if ( s_l > 0 ) iFlx = F_l;
//   else if ( s_c < 0 ) iFlx = F_c;
//   else  iFlx = ( s_c*F_l - s_l*F_c + s_l*s_c*( rho_c - rho_l ) ) / ( s_c - s_l );
//   surf3Dwrite(  iFlx, surf_1,  t_j*sizeof(cudaP), t_i, t_k,  hipBoundaryModeClamp);
//
//   //iFlx rho * vx
//   if ( coord == 1 ){
//     F_l = rho_l * vx_l * vx_l + p_l;
//     F_c = rho_c * vx_c * vx_c + p_c;
//   }
//   else if ( coord == 2 ){
//     F_l = rho_l * vx_l * vy_l;
//     F_c = rho_c * vx_c * vy_c;
//   }
//   else if ( coord == 3 ){
//     F_l = rho_l * vx_l * vz_l;
//     F_c = rho_c * vx_c * vz_c;
//   }
//   if ( s_l > 0 ) iFlx = F_l;
//   else if ( s_c < 0 ) iFlx = F_c;
//   else  iFlx = ( s_c*F_l - s_l*F_c + s_l*s_c*( rho_c*vx_c - rho_l*vx_l ) ) / ( s_c - s_l );
//   surf3Dwrite(  iFlx, surf_2,  t_j*sizeof(cudaP), t_i, t_k,  hipBoundaryModeClamp);
//
//   //iFlx rho * vy
//   if ( coord == 1 ){
//     F_l = rho_l * vy_l * vx_l ;
//     F_c = rho_c * vy_c * vx_c ;
//   }
//   else if ( coord == 2 ){
//     F_l = rho_l * vy_l * vy_l + p_l;
//     F_c = rho_c * vy_c * vy_c + p_c;
//   }
//   else if ( coord == 3 ){
//     F_l = rho_l * vy_l * vz_l;
//     F_c = rho_c * vy_c * vz_c;
//   }
//   if ( s_l > 0 ) iFlx = F_l;
//   else if ( s_c < 0 ) iFlx = F_c;
//   else  iFlx = ( s_c*F_l - s_l*F_c + s_l*s_c*( rho_c*vy_c - rho_l*vy_l ) ) / ( s_c - s_l );
//   surf3Dwrite(  iFlx, surf_3,  t_j*sizeof(cudaP), t_i, t_k,  hipBoundaryModeClamp);
//
//   //iFlx rho * vz
//   if ( coord == 1 ){
//     F_l = rho_l * vz_l * vx_l ;
//     F_c = rho_c * vz_c * vx_c ;
//   }
//   else if ( coord == 2 ){
//     F_l = rho_l * vz_l * vy_l ;
//     F_c = rho_c * vz_c * vy_c ;
//   }
//   else if ( coord == 3 ){
//     F_l = rho_l * vz_l * vz_l + p_l ;
//     F_c = rho_c * vz_c * vz_c + p_c ;
//   }
//   if ( s_l > 0 ) iFlx = F_l;
//   else if ( s_c < 0 ) iFlx = F_c;
//   else  iFlx = ( s_c*F_l - s_l*F_c + s_l*s_c*( rho_c*vz_c - rho_l*vz_l ) ) / ( s_c - s_l );
//   surf3Dwrite(  iFlx, surf_4,  t_j*sizeof(cudaP), t_i, t_k,  hipBoundaryModeClamp);
//
//   //iFlx E
//   if ( coord == 1 ){
//     F_l = vx_l * ( E_l + p_l ) ;
//     F_c = vx_c * ( E_c + p_c ) ;
//   }
//   else if ( coord == 2 ){
//     F_l = vy_l * ( E_l + p_l ) ;
//     F_c = vy_c * ( E_c + p_c ) ;
//   }
//   else if ( coord == 3 ){
//     F_l = vz_l * ( E_l + p_l ) ;
//     F_c = vz_c * ( E_c + p_c ) ;
//   }
//   if ( s_l > 0 ) iFlx = F_l;
//   else if ( s_c < 0 ) iFlx = F_c;
//   else  iFlx = ( s_c*F_l - s_l*F_c + s_l*s_c*( E_c - E_l ) ) / ( s_c - s_l );
//   surf3Dwrite(  iFlx, surf_5,  t_j*sizeof(cudaP), t_i, t_k,  hipBoundaryModeClamp);
//
// }
//
// __global__ void getInterFlux_hll( const int coord, const cudaP dt,  const cudaP gamma, const cudaP dx, const cudaP dy, const cudaP dz,
// 			 cudaP* cnsv_1, cudaP* cnsv_2, cudaP* cnsv_3, cudaP* cnsv_4, cudaP* cnsv_5,
// 			 cudaP* gForceX, cudaP* gForceY, cudaP* gForceZ, cudaP* gravWork ){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int t_k = blockIdx.z*blockDim.z + threadIdx.z;
//   int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//
//   //Read inter-cell fluxes from textures
//
//   cudaP iFlx1_l, iFlx2_l, iFlx3_l, iFlx4_l, iFlx5_l;
//   cudaP iFlx1_r, iFlx2_r, iFlx3_r, iFlx4_r, iFlx5_r;
//   cudaP delta;
//   if ( coord == 1 ){
//     delta = dt / dx;
//     iFlx1_l = fp_tex3D( tex_1, t_j, t_i, t_k);
//     iFlx1_r = fp_tex3D( tex_1, t_j+1, t_i, t_k);
//
//     iFlx2_l = fp_tex3D( tex_2, t_j, t_i, t_k);
//     iFlx2_r = fp_tex3D( tex_2, t_j+1, t_i, t_k);
//
//     iFlx3_l = fp_tex3D( tex_3, t_j, t_i, t_k);
//     iFlx3_r = fp_tex3D( tex_3, t_j+1, t_i, t_k);
//
//     iFlx4_l = fp_tex3D( tex_4, t_j, t_i, t_k);
//     iFlx4_r = fp_tex3D( tex_4, t_j+1, t_i, t_k);
//
//     iFlx5_l = fp_tex3D( tex_5, t_j, t_i, t_k);
//     iFlx5_r = fp_tex3D( tex_5, t_j+1, t_i, t_k);
//   }
//   else if ( coord == 2 ){
//     delta = dt / dy;
//     iFlx1_l = fp_tex3D( tex_1, t_j, t_i, t_k);
//     iFlx1_r = fp_tex3D( tex_1, t_j, t_i+1, t_k);
//
//     iFlx2_l = fp_tex3D( tex_2, t_j, t_i, t_k);
//     iFlx2_r = fp_tex3D( tex_2, t_j, t_i+1, t_k);
//
//     iFlx3_l = fp_tex3D( tex_3, t_j, t_i, t_k);
//     iFlx3_r = fp_tex3D( tex_3, t_j, t_i+1, t_k);
//
//     iFlx4_l = fp_tex3D( tex_4, t_j, t_i, t_k);
//     iFlx4_r = fp_tex3D( tex_4, t_j, t_i+1, t_k);
//
//     iFlx5_l = fp_tex3D( tex_5, t_j, t_i, t_k);
//     iFlx5_r = fp_tex3D( tex_5, t_j, t_i+1, t_k);
//   }
//   else if ( coord == 3 ){
//     delta = dt / dz;
//     iFlx1_l = fp_tex3D( tex_1, t_j, t_i, t_k);
//     iFlx1_r = fp_tex3D( tex_1, t_j, t_i, t_k+1);
//
//     iFlx2_l = fp_tex3D( tex_2, t_j, t_i, t_k);
//     iFlx2_r = fp_tex3D( tex_2, t_j, t_i, t_k+1);
//
//     iFlx3_l = fp_tex3D( tex_3, t_j, t_i, t_k);
//     iFlx3_r = fp_tex3D( tex_3, t_j, t_i, t_k+1);
//
//     iFlx4_l = fp_tex3D( tex_4, t_j, t_i, t_k);
//     iFlx4_r = fp_tex3D( tex_4, t_j, t_i, t_k+1);
//
//     iFlx5_l = fp_tex3D( tex_5, t_j, t_i, t_k);
//     iFlx5_r = fp_tex3D( tex_5, t_j, t_i, t_k+1);
//   }
//
//   //Advance the consv values
//   // cnsv_1[ tid ] = cnsv_1[ tid ] - delta*( iFlx1_r - iFlx1_l );
//   // cnsv_2[ tid ] = cnsv_2[ tid ] - delta*( iFlx2_r - iFlx2_l ) + dt*gForceX[tid]*50;
//   // cnsv_3[ tid ] = cnsv_3[ tid ] - delta*( iFlx3_r - iFlx3_l ) + dt*gForceY[tid]*50;
//   // cnsv_4[ tid ] = cnsv_4[ tid ] - delta*( iFlx4_r - iFlx4_l ) + dt*gForceZ[tid]*50;
//   // cnsv_5[ tid ] = cnsv_5[ tid ] - delta*( iFlx5_r - iFlx5_l ) + dt*gravWork[tid]*50;
//
//   cnsv_1[ tid ] = cnsv_1[ tid ] - delta*( iFlx1_r - iFlx1_l );
//   cnsv_2[ tid ] = cnsv_2[ tid ] - delta*( iFlx2_r - iFlx2_l );
//   cnsv_3[ tid ] = cnsv_3[ tid ] - delta*( iFlx3_r - iFlx3_l );
//   cnsv_4[ tid ] = cnsv_4[ tid ] - delta*( iFlx4_r - iFlx4_l );
//   cnsv_5[ tid ] = cnsv_5[ tid ] - delta*( iFlx5_r - iFlx5_l );
// }
//
//
// __global__ void iterPoissonStep( int* converged, const int paridad,
// 				 const int nWidth, const cudaP omega, const cudaP pi4,
// 				 cudaP dx, cudaP dy, cudaP dz,
// 				 cudaP* rhoVals, cudaP* phiVals, float* phiWall ){
//   int t_j = 2*(blockIdx.x*blockDim.x + threadIdx.x);
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int t_k = blockIdx.z*blockDim.z + threadIdx.z;
//   //Make a checkboard 3D grid
//   if ( t_i%2 == 0 ){
//     if ( t_k%2 == paridad ) t_j +=1;
//   }
//   else if ( (t_k+1)%2 == paridad ) t_j +=1;
//   int tid = t_j + t_i*nWidth + t_k*nWidth*blockDim.y*gridDim.y;
//
//   cudaP rho, phi_c, phi_l, phi_r, phi_d, phi_u, phi_b, phi_t, phi_new;
//   rho = rhoVals[ tid ];
//   phi_c = fp_tex3D( tex_1, t_j, t_i, t_k);
//   phi_l = fp_tex3D( tex_1, t_j-1, t_i, t_k);
//   phi_r = fp_tex3D( tex_1, t_j+1, t_i, t_k);
//   phi_d = fp_tex3D( tex_1, t_j, t_i-1, t_k);
//   phi_u = fp_tex3D( tex_1, t_j, t_i+1, t_k);
//   phi_b = fp_tex3D( tex_1, t_j, t_i, t_k-1);
//   phi_t = fp_tex3D( tex_1, t_j, t_i, t_k+1);
//
//   //Boundary conditions
//   if  ( t_j == 0 )        phi_l = phi_r;
//   if  ( t_j == nWidth-1 ) phi_r = phi_l;
//   if  ( t_i == 0 )        phi_d = phi_u;
//   if  ( t_i == nWidth-1 ) phi_u = phi_d;
//   if  ( t_k == 0 )        phi_b = phi_t;
//   if  ( t_k == nWidth-1 ) phi_t = phi_b;
//
// //   phi_new =  1./6 * ( phi_l + phi_r + phi_d + phi_u + phi_b + phi_t - dx*dx*rho   );
//   phi_new = (1-omega)*phi_c + omega/6*( phi_l + phi_r + phi_d + phi_u + phi_b + phi_t - dx*dx*pi4*rho );
//
//   if ( paridad == 0 ) surf3Dwrite(  phi_new, surf_1,  t_j*sizeof(cudaP), t_i, t_k,  hipBoundaryModeClamp);
//   phiVals[ tid ] = phi_new;
//
// //   if ( ( t_j == 0 ) ||  ( t_j == nWidth-1 ) || ( t_i == 0 ) || ( t_i == nWidth-1 ) || ( t_k == 0 ) || ( t_k == nWidth-1 ) ) return;
// //   if ( ( blockIdx.x == 0 ) ||  ( blo == nWidth-1 ) || ( t_i == 0 ) || ( t_i == nWidth-1 ) || ( t_k == 0 ) || ( t_k == nWidth-1 ) ) return;
//
//   if ( ( abs( ( phi_new - phi_c ) / phi_c ) > 0.002 ) ) converged[0] = 0;
//
//
// }
//
// __global__ void getGravityForce( const int nWidth, const int nHeight, const int nDepth,
// 				 cudaP dx, cudaP dy, cudaP dz,
// 				 cudaP* gForce_x, cudaP* gForce_y, cudaP* gForce_z,
// 				 cudaP* rho, cudaP* pX, cudaP* pY, cudaP* pZ, cudaP *gravWork,
// 				 float* phiWall      ){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int t_k = blockIdx.z*blockDim.z + threadIdx.z;
//   int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//
//   cudaP phi_l, phi_r, phi_d, phi_u, phi_b, phi_t;
// //   phi_c = fp_tex3D( tex_1, t_j, t_i, t_k);
//   phi_l = fp_tex3D( tex_1, t_j-1, t_i, t_k);
//   phi_r = fp_tex3D( tex_1, t_j+1, t_i, t_k);
//   phi_d = fp_tex3D( tex_1, t_j, t_i-1, t_k);
//   phi_u = fp_tex3D( tex_1, t_j, t_i+1, t_k);
//   phi_b = fp_tex3D( tex_1, t_j, t_i, t_k-1);
//   phi_t = fp_tex3D( tex_1, t_j, t_i, t_k+1);
//
//   //Boundary conditions
//   if  ( t_j == 0 )        phi_l = phi_r;
//   if  ( t_j == nWidth-1 ) phi_r = phi_l;
//   if  ( t_i == 0 )        phi_d = phi_u;
//   if  ( t_i == nWidth-1 ) phi_u = phi_d;
//   if  ( t_k == 0 )        phi_b = phi_t;
//   if  ( t_k == nWidth-1 ) phi_t = phi_b;
//
//   //Get partial derivatives for force
//   cudaP gField_x, gField_y, gField_z, p_x, p_y, p_z, rho_c;
//   rho_c = rho[ tid ];
//   gField_x = ( phi_l - phi_r ) * 0.5 / dx;
//   gField_y = ( phi_d - phi_u ) * 0.5 / dy;
//   gField_z = ( phi_b - phi_t ) * 0.5 / dz;
//   gForce_x[ tid ] = gField_x * rho_c;
//   gForce_y[ tid ] = gField_y * rho_c;
//   gForce_z[ tid ] = gField_z * rho_c;
// //   gForce_x[ tid ] = gField_x;
// //   gForce_y[ tid ] = gField_y;
// //   gForce_z[ tid ] = gField_z;
//
//   //Get momentum for virtual gravitational work
//   p_x = pX[ tid ] ;
//   p_y = pY[ tid ] ;
//   p_z = pZ[ tid ] ;
//   gravWork[ tid ] = p_x * gField_x + p_y * gField_y + p_z * gField_z ;
//
// }
//
// __global__ void reduceDensity( const int nWidth, const int nHeight, const int nDepth,
// 			       const float dx, const float dy, const float dz,
// 			       const float xMin, const float yMin, const float zMin,
// 			       cudaP* rhoAll, float* rhoReduced,
// 			       float* blockX, float* blockY, float* blockZ  ){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int t_k = blockIdx.z*blockDim.z + threadIdx.z;
//   int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   int tid_b = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
//   int bid = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
//
//   float rho = float( rhoAll[tid] );
//   __shared__ float density[ THREADS_PER_BLOCK ];
//
//   density[tid_b] = rho;
//   __syncthreads();
//
//   int i =  blockDim.x * blockDim.y * blockDim.z  / 2;
//   while ( i > 0 ){
//     if ( tid_b < i ) density[tid_b] = density[tid_b] + density[tid_b+i];
//     __syncthreads();
//     i /= 2;
//   }
//
//
//   float x = blockDim.x*dx * ( blockIdx.x + 0.5f ) + xMin;
//   float y = blockDim.y*dy * ( blockIdx.y + 0.5f ) + yMin;
//   float z = blockDim.z*dz * ( blockIdx.z + 0.5f ) + zMin;
//   if (tid_b == 0 ){
//     rhoReduced[ bid ] = density[0]*dx*dy*dz ;
//     blockX[ bid ] = x;
//     blockY[ bid ] = y;
//     blockZ[ bid ] = z;
//   }
//
// }
// __global__ void getBounderyPotential(const float pi4, const int nBlocks, const int nWidth, const int nHeight, const int nDepth,
// 			      float dx, float dy, float dz, float xMin, float yMin, float zMin,
// 			      float* rhoReduced,  float* phiWall,
// 			      float* blockX, float* blockY, float* blockZ   ){
// // 			      float* phiWall_l, float* phiWall_r, float* phiWall_d, float* phiWall_u, float* phiWall_b, float* phiWall_t){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int tid = t_j + t_i*blockDim.x*gridDim.x ;
// //   int tid_b = threadIdx.x + threadIdx.y*blockDim.x
// //   int bid = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
//
//   float y_wall = yMin + t_j*dy;
//   float z_wall = zMin + t_i*dz;
//
//   float x_b, y_b, z_b, phi, rho ;
//   phi = 0;
//
//   for ( int nBlock = 0; nBlock<nBlocks; nBlock++ ){
//     rho = rhoReduced[ nBlock ];
//     x_b = blockX[nBlock];
//     y_b = blockY[nBlock];
//     z_b = blockZ[nBlock];
//     phi -= rsqrt( x_b*x_b + (y_b-y_wall)*(y_b-y_wall) + (z_b-z_wall)*(z_b-z_wall) ) * rho;
//   }
//   phiWall[ tid ] = phi;
//
// }
//
//
//
// // __global__ void getBounderyPotential( const int nWidth, const int nHeight, const int nDepth,
// // 			      float dx, float dy, float dz, float xMin, float yMin, float zMin,
// // 			      cudaP* rhoAll, float* phiWall ){
// // // 			      float* phiWall_l, float* phiWall_r, float* phiWall_d, float* phiWall_u, float* phiWall_b, float* phiWall_t){
// //   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
// //   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
// //   int t_k = blockIdx.z*blockDim.z + threadIdx.z;
// //   int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
// //   int tid_b = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
// //
// //   const int ny = 8*8*2;
// //   const int nz = 2;
// //
// //   const int nSwipes = 50;
// //
// //   float rho = float ( rhoAll[ tid ] );
// //
// //   float x, y, z, y_wall, z_wall, phi;
// //   x = xMin + t_j*dx;
// //   y = yMin + t_i*dy;
// //   z = zMin + t_k*dz;
// //
// //   int idx_1, idx_2, i, j, swipeCounter, swipeIdx;
// //   swipeIdx = tid / 256;
// //
// //   //Allocate shared memory
// //   __shared__ float wallStripe[ 256 ];
// //
// //   for ( swipeCounter=0; swipeCounter<nSwipes; swipeCounter++ ){
// //     //Initialize shared memory to zero
// //     wallStripe[ tid_b ] = 0;
// //     __syncthreads();
// //
// //     //Initialize the indexes over the tile
// //     idx_1 = tid_b % ny;
// //     idx_2 = tid_b / ny;
// //
// //     //Fill the tile of the wall
// //     for ( j=0; j<nz; j++ ){
// //       z_wall = idx_2*dz + zMin;
// //       for ( i=0; i<ny; i++ ){
// // 	y_wall = idx_1*dy + yMin;
// // 	phi = rsqrt( x*x + (y-y_wall)*(y_wall) + (z-z_wall)*(z-z_wall) ) * rho;
// // 	wallStripe[ idx_2*ny + idx_1 ] += phi;
// // 	idx_1 += 1;
// // 	if ( idx_1 >= ny ) idx_1 = 0;
// //       }
// //       idx_2 += 1;
// //       if ( idx_2 >= nz ) idx_2 = 0;
// //     }
// //
// //     //Write the tile values to global memory
// //     idx_1 = tid_b % ny;
// //     idx_2 = tid_b / ny;
// //     atomicAdd( &phiWall[ swipeIdx*256 + idx_2*ny + idx_1  ], wallStripe[ tid_b ] ) ;
// // //     swipeIdx += 1;
// // //     if ( swipeIdx >= 128 ) swipeIdx = 0;
// //   }
// //
// // }
